#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#define tpb 128

using namespace cv;

__global__ void Sharpen(unsigned char* in, unsigned char* out, int total_pixels, int w, int down, int up) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i > down && i < up){
    if (i%w !=0 && i%w != w-1){
      //find nearby positions
      int up = i - 3*w;
      int down = i + 3*w;
      int left = i - 3;
      int right = i + 3;

      int tmp = 5*in[i]-in[up]-in[down]-in[left]-in[right];
      if(tmp<0) tmp=0;
      if(tmp>255) tmp=255;

      out[i]=tmp;

      //out[i]=in[i];
    }
    else {
      out[i]=in[i];
    }
  }
  else {
    out[i]=in[i];
  }
  
}


int main()
{

	Mat img = imread("gs.jpg",IMREAD_COLOR);
	Size s = img.size();
	int w = s.width;
	int h = s.height;

	Mat img_invert(h, w, CV_8UC3, Scalar(0,0,0));

	unsigned char* char_img = img.data;
	unsigned char* new_img = img_invert.data;

	int u_char_size = h * w * 3 * sizeof(unsigned char);

	unsigned char *ar_img, *ar_img_inv;

	int vec_size = h * w * 3;
	int block_count = ((vec_size + tpb - 1)/tpb) + 1;

	hipMalloc((void**) &ar_img, u_char_size);
	hipMalloc((void**) &ar_img_inv, u_char_size);

	hipMemcpy(ar_img, char_img, u_char_size, hipMemcpyHostToDevice);
	hipMemcpy(ar_img_inv, new_img, u_char_size, hipMemcpyHostToDevice);

  int down = 3*w;
  int up = vec_size - 3*w;

	Sharpen<<<block_count, tpb>>>  (ar_img, ar_img_inv, vec_size, w, down, up);

	hipMemcpy(char_img, ar_img, u_char_size, hipMemcpyDeviceToHost);
	hipMemcpy(new_img, ar_img_inv, u_char_size, hipMemcpyDeviceToHost);

	hipFree(ar_img);
	hipFree(ar_img_inv);
   
	Mat output = Mat(h, w, CV_8UC3, new_img);
	imwrite("sharp.jpg", output);
}